#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>

// nvcc -rdc=true -ccbin g++ -arch=sm_90 hello_nvshmem.cu -lnvshmem -lnvidia-ml -lcuda -lcudart -I/workspace/byte-flux/3rdparty/nvshmem/build/src/include -L/workspace/byte-flux/3rdparty/nvshmem/build/src/lib
// find / -name nvshmem_bootstrap_pmi.so.3
// export LD_LIBRARY_PATH=/workspace/byte-flux/3rdparty/nvshmem/build/src/lib:$LD_LIBRARY_PATH

// install nvshmrun: /workspace/byte-flux/3rdparty/nvshmem/scripts/install_nvshmrun.sh
// nvshrun xx

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(void) {
    int mype_node, msg;
    hipStream_t stream;

    nvshmem_init();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    int *destination = (int *) nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    return 0;
}
